#include "hip/hip_runtime.h"
#include "PCFG.h"
#include <hip/hip_runtime.h>
#include <>
#include <vector>
#include <string>
#include <iostream>
#include <cstring>

using namespace std;

// ==================== 常量定义区域 ====================
// GPU加速处理的数据量阈值
#define GPU_ACCELERATION_THRESHOLD 1000

// CUDA运行时错误检测宏定义
#define CHECK_CUDA_ERROR(operation) do { \
    hipError_t cuda_result = operation; \
    if (cuda_result != hipSuccess) { \
        fprintf(stderr, "CUDA运行错误 %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(cuda_result)); \
        exit(1); \
    } \
} while(0)

// ==================== 前置声明区域 ====================
class CudaComputeManager;
static CudaComputeManager* global_cuda_manager = nullptr;

// ==================== 核函数定义区域 ====================
// CUDA核心函数：字符串拼接处理
__global__ void string_concatenation_kernel(const char* input_data, const int* data_offsets, const int* data_lengths,
                                          const char* prefix_data, int prefix_length,
                                          char* output_buffer, int max_string_length, int total_items) {
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id >= total_items) return;

    char* current_output = output_buffer + thread_id * max_string_length;
    int current_position = 0;

    // 拷贝前缀字符串
    if (prefix_data && prefix_length > 0) {
        for (int i = 0; i < prefix_length && current_position < max_string_length - 1; i++) {
            current_output[current_position++] = prefix_data[i];
        }
    }

    // 拷贝主要数据
    const char* source_data = input_data + data_offsets[thread_id];
    int source_length = data_lengths[thread_id];
    for (int i = 0; i < source_length && current_position < max_string_length - 1; i++) {
        current_output[current_position++] = source_data[i];
    }

    current_output[current_position] = '\0';
}

// ==================== 工具函数区域 ====================
// GPU计算环境初始化
void initialize_cuda_environment();
void release_cuda_resources();
bool processWithGPU(const vector<string>& values, const string& prefix, vector<string>& results);

// ==================== 核心类定义区域 ====================
// GPU计算资源管理器
class CudaComputeManager {
private:
    // 内存容量限制常量
    static const size_t MAXIMUM_ITEM_COUNT = 120000;
    static const size_t MAXIMUM_INPUT_CAPACITY = 60 * 1024 * 1024;  // 50MB
    static const size_t MAXIMUM_OUTPUT_CAPACITY = 120 * 1024 * 1024; // 100MB

    // CUDA设备内存指针
    char* cuda_input_buffer = nullptr;
    char* cuda_prefix_buffer = nullptr;
    char* cuda_result_buffer = nullptr;
    int* cuda_offset_array = nullptr;
    int* cuda_length_array = nullptr;
    hipStream_t hip_stream;

public:
    CudaComputeManager() {
        // 分配GPU内存空间
        CHECK_CUDA_ERROR(hipMalloc(&cuda_input_buffer, MAXIMUM_INPUT_CAPACITY));
        CHECK_CUDA_ERROR(hipMalloc(&cuda_prefix_buffer, 1024)); // 前缀缓冲区1KB
        CHECK_CUDA_ERROR(hipMalloc(&cuda_result_buffer, MAXIMUM_OUTPUT_CAPACITY));
        CHECK_CUDA_ERROR(hipMalloc(&cuda_offset_array, MAXIMUM_ITEM_COUNT * sizeof(int)));
        CHECK_CUDA_ERROR(hipMalloc(&cuda_length_array, MAXIMUM_ITEM_COUNT * sizeof(int)));
        CHECK_CUDA_ERROR(hipStreamCreate(&hip_stream));
    }

    ~CudaComputeManager() {
        // 释放GPU内存资源
        hipFree(cuda_input_buffer);
        hipFree(cuda_prefix_buffer);
        hipFree(cuda_result_buffer);
        hipFree(cuda_offset_array);
        hipFree(cuda_length_array);
        hipStreamDestroy(hip_stream);
    }

    // 批量处理字符串拼接任务
    bool process_string_batch(const vector<string>& input_strings, const string& prefix_string, vector<string>& output_results) {
        size_t string_count = input_strings.size();
        if (string_count == 0 || string_count > MAXIMUM_ITEM_COUNT) return false;

        // 计算输入数据总大小
        size_t total_input_bytes = 0;
        for(const auto& str : input_strings) {
            total_input_bytes += str.length();
        }
        if (total_input_bytes > MAXIMUM_INPUT_CAPACITY) return false;

        char* host_input_data = (char*)malloc(total_input_bytes);
        vector<int> host_offset_data(string_count);
        vector<int> host_length_data(string_count);
        if (!host_input_data) return false;

        size_t byte_position = 0;
        for(size_t i = 0; i < string_count; ++i) {
            host_offset_data[i] = byte_position;
            host_length_data[i] = input_strings[i].length();
            memcpy(host_input_data + byte_position, input_strings[i].c_str(), host_length_data[i]);
            byte_position += host_length_data[i];
        }

        const int MAXIMUM_STRING_LENGTH = 64;
        size_t result_buffer_size = string_count * MAXIMUM_STRING_LENGTH;
        if (result_buffer_size > MAXIMUM_OUTPUT_CAPACITY) {
            free(host_input_data);
            return false;
        }

        char* host_result_data = (char*)malloc(result_buffer_size);
        if (!host_result_data) {
            free(host_input_data);
            return false;
        }

        // 异步数据传输：主机到设备
        CHECK_CUDA_ERROR(hipMemcpyAsync(cuda_input_buffer, host_input_data, total_input_bytes, hipMemcpyHostToDevice, hip_stream));
        CHECK_CUDA_ERROR(hipMemcpyAsync(cuda_offset_array, host_offset_data.data(), string_count * sizeof(int), hipMemcpyHostToDevice, hip_stream));
        CHECK_CUDA_ERROR(hipMemcpyAsync(cuda_length_array, host_length_data.data(), string_count * sizeof(int), hipMemcpyHostToDevice, hip_stream));

        char* device_prefix_pointer = nullptr;
        if (!prefix_string.empty()) {
            CHECK_CUDA_ERROR(hipMemcpyAsync(cuda_prefix_buffer, prefix_string.c_str(), prefix_string.length(), hipMemcpyHostToDevice, hip_stream));
            device_prefix_pointer = cuda_prefix_buffer;
        }

        // 启动CUDA核函数
        const int threads_per_block = 512;
        int blocks_per_grid = (string_count + threads_per_block - 1) / threads_per_block;
        string_concatenation_kernel<<<blocks_per_grid, threads_per_block, 0, hip_stream>>>(
            cuda_input_buffer, cuda_offset_array, cuda_length_array, device_prefix_pointer, prefix_string.length(),
            cuda_result_buffer, MAXIMUM_STRING_LENGTH, string_count);

        // 异步数据传输：设备到主机
        CHECK_CUDA_ERROR(hipMemcpyAsync(host_result_data, cuda_result_buffer, result_buffer_size, hipMemcpyDeviceToHost, hip_stream));

        // 同步等待计算完成
        CHECK_CUDA_ERROR(hipStreamSynchronize(hip_stream));

        output_results.reserve(output_results.size() + string_count);
        for (size_t i = 0; i < string_count; i++) {
            output_results.emplace_back(host_result_data + i * MAXIMUM_STRING_LENGTH);
        }

        // 清理主机内存
        free(host_input_data);
        free(host_result_data);

        return true;
    }
};

// ==================== 接口函数实现区域 ====================
void initialize_cuda_environment() {
    if (!global_cuda_manager) {
        global_cuda_manager = new CudaComputeManager();
    }
}

void cleanup_gpu_resources() {
    if (global_cuda_manager) {
        delete global_cuda_manager;
        global_cuda_manager = nullptr;
    }
}

// 对外提供的GPU处理接口
bool processWithGPU(const vector<string>& values, const string& prefix, vector<string>& results) {
    if (!global_cuda_manager) {
        initialize_cuda_environment();
    }
    return global_cuda_manager->process_string_batch(values, prefix, results);
}

// ==================== PCFG算法实现区域 ====================
// 以下代码保持不变（L189-360）
void PriorityQueue::CalProb(PT &pt)
{
    pt.prob = pt.preterm_prob;
    int index = 0;

    for (int idx : pt.curr_indices)
    {
        if (pt.content[index].type == 1)
        {
            pt.prob *= m.letters[m.FindLetter(pt.content[index])].ordered_freqs[idx];
            pt.prob /= m.letters[m.FindLetter(pt.content[index])].total_freq;
        }
        if (pt.content[index].type == 2)
        {
            pt.prob *= m.digits[m.FindDigit(pt.content[index])].ordered_freqs[idx];
            pt.prob /= m.digits[m.FindDigit(pt.content[index])].total_freq;
        }
        if (pt.content[index].type == 3)
        {
            pt.prob *= m.symbols[m.FindSymbol(pt.content[index])].ordered_freqs[idx];
            pt.prob /= m.symbols[m.FindSymbol(pt.content[index])].total_freq;
        }
        index += 1;
    }
}

void PriorityQueue::init()
{
    initialize_cuda_environment();
    for (PT pt : m.ordered_pts)
    {
        for (segment seg : pt.content)
        {
            if (seg.type == 1)
            {
                pt.max_indices.emplace_back(m.letters[m.FindLetter(seg)].ordered_values.size());
            }
            if (seg.type == 2)
            {
                pt.max_indices.emplace_back(m.digits[m.FindDigit(seg)].ordered_values.size());
            }
            if (seg.type == 3)
            {
                pt.max_indices.emplace_back(m.symbols[m.FindSymbol(seg)].ordered_values.size());
            }
        }
        pt.preterm_prob = float(m.preterm_freq[m.FindPT(pt)]) / m.total_preterm;
        CalProb(pt);
        priority.emplace_back(pt);
    }
}

void PriorityQueue::PopNext()
{
    const int BATCH_SIZE = 64;// 批处理大小
    vector<PT> batch_pts;

    for (int i = 0; i < BATCH_SIZE && !priority.empty(); i++) {
        batch_pts.push_back(priority.front());
        priority.erase(priority.begin());
    }

    if (!batch_pts.empty()) {
        BatchGenerate(batch_pts);
    }

    for (PT& processed_pt : batch_pts) {
        vector<PT> new_pts = processed_pt.NewPTs();
        for (PT pt : new_pts) {
            CalProb(pt);
            bool inserted = false;
            for (auto iter = priority.begin(); iter != priority.end(); iter++) {
                if (pt.prob > iter->prob) {
                    priority.emplace(iter, pt);
                    inserted = true;
                    break;
                }
            }
            if (!inserted) {
                priority.emplace_back(pt);
            }
        }
    }
}

vector<PT> PT::NewPTs()
{
    vector<PT> res;
    if (content.size() == 1) {
        return res;
    }
    else {
        int init_pivot = pivot;
        for (int i = pivot; i < curr_indices.size() - 1; i += 1) {
            curr_indices[i] += 1;
            if (curr_indices[i] < max_indices[i]) {
                pivot = i;
                res.emplace_back(*this);
            }
            curr_indices[i] -= 1;
        }
        pivot = init_pivot;
        return res;
    }
}

void PriorityQueue::Generate(PT pt)
{
    CalProb(pt);

    if (pt.content.size() == 1)
    {
        segment *a;
        if (pt.content[0].type == 1) a = &m.letters[m.FindLetter(pt.content[0])];
        if (pt.content[0].type == 2) a = &m.digits[m.FindDigit(pt.content[0])];
        if (pt.content[0].type == 3) a = &m.symbols[m.FindSymbol(pt.content[0])];

        int num_values = pt.max_indices[0];

        if (num_values >= GPU_ACCELERATION_THRESHOLD) {
            vector<string> temp_results;
            if (processWithGPU(a->ordered_values, "", temp_results)) {
                guesses.insert(guesses.end(), temp_results.begin(), temp_results.end());
                total_guesses += temp_results.size();
                return;
            }
        }

        // GPU处理失败或数据量不足，使用CPU处理
        for (int i = 0; i < num_values; i++)
        {
            guesses.push_back(a->ordered_values[i]);
            total_guesses++;
        }
    }
    else
    {
        string prefix;
        int seg_idx = 0;
        for (int idx : pt.curr_indices)
        {
            if (pt.content[seg_idx].type == 1) prefix += m.letters[m.FindLetter(pt.content[seg_idx])].ordered_values[idx];
            if (pt.content[seg_idx].type == 2) prefix += m.digits[m.FindDigit(pt.content[seg_idx])].ordered_values[idx];
            if (pt.content[seg_idx].type == 3) prefix += m.symbols[m.FindSymbol(pt.content[seg_idx])].ordered_values[idx];
            seg_idx += 1;
            if (seg_idx == pt.content.size() - 1) break;
        }

        segment *a;
        if (pt.content.back().type == 1) a = &m.letters[m.FindLetter(pt.content.back())];
        if (pt.content.back().type == 2) a = &m.digits[m.FindDigit(pt.content.back())];
        if (pt.content.back().type == 3) a = &m.symbols[m.FindSymbol(pt.content.back())];

        int num_values = pt.max_indices.back();

        if (num_values >= GPU_ACCELERATION_THRESHOLD) {
            vector<string> temp_results;
            if (processWithGPU(a->ordered_values, prefix, temp_results)) {
                guesses.insert(guesses.end(), temp_results.begin(), temp_results.end());
                total_guesses += temp_results.size();
                return;
            }
        }

        // GPU处理失败或数据量不足，使用CPU处理
        for (int i = 0; i < num_values; i++)
        {
            guesses.push_back(prefix + a->ordered_values[i]);
            total_guesses++;
        }
    }
}

void PriorityQueue::BatchGenerate(const vector<PT>& batch_pts) {
    for (const PT& pt : batch_pts) {
        Generate(const_cast<PT&>(pt));
    }
}